#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>
#include <cmath>

#include "primitives/Vec3.cuh"
#include "utils/utils.cuh"
#include "utils/definitions.cuh"
#include "IO/Raster.h"
#include "bvh/BVH.cuh"
#include "array/Array.cuh"
#include "primitives/Point3.cuh"
#include "primitives/Ray.cuh"


/*__global__
void buildBVH(BVH<float>** bvh, unsigned int nbPixels){
    &bvh = new BVH<float>(*pointsArray, BVHNodeMemory, bboxMemory, elementsMemory);
    bvh->build();
}*/

__global__ 
void initRender(int maxX, int maxY, hiprandState* randomState) {
   const int x = threadIdx.x + blockIdx.x * blockDim.x;
   const int y = threadIdx.y + blockIdx.y * blockDim.y;
   if(x>=maxX || y>=maxY) return;
   const int index = y*maxX + x;
   hiprand_init(1423, index, 0, &randomState[index]);
}


__global__
void trace(float* data, Point3<float>* points, int maxX, int maxY, BVH<float>* bvh, const int raysPerPoint, hiprandState* randomState, BVHNode<float>** traceBuffer, int traceBufferSize, int nbDirs){
    /*const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x>=maxX || y>=maxY) return;
    const int index = y*maxX + x;

    hiprandState localRndState = randomState[index];

    Point3<float> origin  = points[index];
    Vec3<float> direction = Vec3<float>(0,0,0);
    Ray<float> ray        = Ray<float>(origin, direction);

    float result = 0;
    for(int i=0; i<raysPerPoint; i++){
        const float p = ray.getDirection().setRandomInHemisphereCosine(localRndState, nbDirs, i%nbDirs);
        result += bvh->getLighting(ray, &traceBuffer[index*traceBufferSize])/p;
    }
    data[index] = result/raysPerPoint;*/
}

int main(){
    const bool USE_GPU = false;
    const bool PRINT_INFOS = true;
    const char* filename = "data/input.tif";
    const char* outputFilename = "data/output.tif";

    Raster raster = Raster(filename, outputFilename);
    const unsigned int nbPixels = raster.getWidth()*raster.getHeight();

    if(PRINT_INFOS){
        raster.printInfos();
        printDevicesInfos();     
    }

    if(USE_GPU){

    }else{

    }

    // Read data from raster
    float* data;
    checkError(hipMallocManaged(&data, nbPixels*sizeof(float)));
    raster.readData(data);

    // Create points in 3D
    Point3<float>* points = (Point3<float>*) allocGPU(nbPixels, sizeof(Point3<float>));

    Point3<float>** pointsArrayContent;
    checkError(hipMallocManaged(&pointsArrayContent, nbPixels*sizeof(Point3<float>*)));

    Point3<float>* pointsArrayMemory;
    checkError(hipMallocManaged(&pointsArrayMemory, sizeof(Array<Point3<float>*>)));
    Array<Point3<float>*>* pointsArray = new (pointsArrayMemory) Array<Point3<float>*>(pointsArrayContent, nbPixels);
    

    for(int y=0; y<raster.getHeight(); y++){
        for(int x=0; x<raster.getWidth(); x++){
            const int index = y*raster.getWidth()+x;
            points[index] = Point3<float>((float)x/2,(float)y/2,data[index]);
            (*pointsArray)[index] = &(points[index]);
        }
    }


    std::cout << "Building BVH...\n";

    BVH<float> bvh = BVH<float>(nbPixels);
    bvh.build(*pointsArray);
    bvh.printInfos();
    
    std::cout << "BVH built\n";


    // Trace
    constexpr unsigned int RAYS_PER_POINT = 64;
    constexpr int NB_STRATIFIED_DIRS = 32;

    const int traceBufferSizePerThread = std::log2(bvh.size())+1;
    BVHNode<float>** traceBuffer = (BVHNode<float>**)allocGPU(nbPixels*traceBufferSizePerThread, sizeof(BVHNode<float>*));

    std::cout << "Start tracing...\n";

    if(USE_GPU){
        /*const dim3 threads(8,8);
        const dim3 blocks(raster.getWidth()/threads.x+1, raster.getHeight()/threads.y+1);
        
        hiprandState* randomState;
        checkError(hipMallocManaged((void **)& randomState, nbPixels*sizeof(hiprandState)));
        
        std::cout << "Initializing tracing...\n";

        initRender<<<blocks, threads>>>(raster.getWidth(), raster.getHeight(), randomState);
        checkError(hipGetLastError());
        checkError(hipDeviceSynchronize());

        std::cout << "Start tracing...\n";

        trace<<<blocks, threads>>>(
            data, points, raster.getWidth(), raster.getHeight(), 
            bvh, RAYS_PER_POINT, randomState, traceBuffer, traceBufferSizePerThread, NB_STRATIFIED_DIRS);
        checkError(hipGetLastError());
        checkError(hipDeviceSynchronize());*/

    }else{

        float progress = 0;
        float nextProgress = 0.1;

        #pragma omp parallel for
        for(int y=0; y<raster.getHeight(); y++){
            for(int x=0; x<raster.getWidth(); x++){
                const int index = y*raster.getWidth() + x;

                Point3<float> origin  = points[index];
                Vec3<float> direction = Vec3<float>(0,0,0);
                Ray<float> ray        = Ray<float>(origin, direction);

                float result = 0;
                for(int i=0; i<RAYS_PER_POINT; i++){
                    const float cosThetaOverP = ray.getDirection().setRandomInHemisphereCosine( NB_STRATIFIED_DIRS , i%NB_STRATIFIED_DIRS );
                    result += cosThetaOverP*bvh.getLighting(ray, &traceBuffer[index*traceBufferSizePerThread]);
                }
                data[index] = result/RAYS_PER_POINT;
            }

            #pragma omp atomic
            progress++;

            if(progress >= nextProgress*raster.getHeight()){
                std::cout << "Progress " << 100*nextProgress << "%\n";
                std::flush(std::cout);
                nextProgress += 0.1;
            }
        }
    }

    std::cout << "Tracing finished...\n";

    raster.writeData(data);

    hipFree(data);

    std::cout << "Finished \n";
    return 0;
}