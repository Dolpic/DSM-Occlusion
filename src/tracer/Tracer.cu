#include "hip/hip_runtime.h"
#include "Tracer.cuh"

#include ""

#include "../utils/utils.cuh"
#include "../array/Array.cuh"

#include <iostream>
#include <random>

std::default_random_engine genEngine;
std::uniform_real_distribution<> uniform0_1 = std::uniform_real_distribution<>(0.001, 1); // Not starting at zero to avoid dividing by zero

constexpr byte NB_STRATIFIED_DIRS = 64; // TODO properly compute this number so that the render is not biased
constexpr uint SEED = 1423; // For reproducible runs, can be any value
constexpr uint BLOCK_DIM_SIZE = 8;
__global__
void renderGPU(const Array2D<float>& data, const Array2D<Point3<float>>& points, const BVH& bvh, const uint raysPerPoint, hiprandState* const rndState){    
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x>=data.width() || y>=data.height()) return;
    const uint index = y*data.width() + x;

    extern __shared__ float sharedMem[];
    //BVHNode* const cache = (BVHNode*) &sharedMem[threads.x*threads.y];

    /*for (int i = 0; i<raysPerPoint / (BLOCK_DIM_SIZE * BLOCK_DIM_SIZE); i++) {
        const int curIndex = (i+1)*(threadIdx.x + threadIdx.y * BLOCK_DIM_SIZE);
        sharedMem[curIndex] = (float)curIndex/raysPerPoint;
    }*/

    hiprandState localRndState = rndState[index];
    hiprand_init(SEED, index, 0, &localRndState);

    const Point3<float> origin(points[index].x, points[index].y, points[index].z);
    Vec3<float> direction = Vec3<float>(0.0, 0.0, 0.0);
    
    __syncthreads(); // Wait for each thread to initialize its part of the shared memory

    float result = 0;
    for(unsigned short i=0; i<raysPerPoint; i++){
        //const float rndPhi = fminf( sharedMem[i] + 0.005*hiprand_uniform(&localRndState), 1);
        const float rndPhi = hiprand_uniform(&localRndState);
        // const float rndTheta = sharedMem[i+1];
        const float rndTheta = hiprand_uniform(&localRndState);
        const float cosThetaOverPdf = direction.setRandomInHemisphereCosineGPU(NB_STRATIFIED_DIRS, raysPerPoint, i, rndPhi, rndTheta);
        const Vec3<float> invDir(fdividef(1,direction.x), fdividef(1,direction.y), fdividef(1,direction.z));
        result += cosThetaOverPdf*bvh.getLighting(origin, invDir);
    }
    data[index] = ONE_OVER_PI*result/raysPerPoint; // Diffuse BSDF
}

Tracer::Tracer(Array2D<float>& data, const float pixelSize): 
    data(data), pixelSize(pixelSize), 
    points(Array2D<Point3<float>>(data.width(), data.height())), 
    bvh(BVH(data.width()*data.height(), pixelSize)){}

Tracer::~Tracer(){
    hipFree(randomState);
    bvh.freeAllMemory();
}

void Tracer::init(const bool prinInfos){
    randomState = (hiprandState*) allocGPU(sizeof(hiprandState), data.width()*data.height());
    Array2D<Point3<float>*> pointsPointers(data.width(), data.height());

    for(uint y=0; y<data.height(); y++){
        for(uint x=0; x<data.width(); x++){
            const uint index = y*data.width()+x;
            points[index] = Point3<float>(x*pixelSize,y*pixelSize, data[index]);
            pointsPointers[index] = &(points[index]);
        }
    }
    bvh.build(pointsPointers);

    if(prinInfos){
        bvh.printInfos();
    }
    bvh.freeAfterBuild();
}

void Tracer::trace(const bool useGPU, const uint raysPerPoint){
    const uint traceBufferSizePerThread = std::log2(bvh.size());
    const dim3 blockDims(BLOCK_DIM_SIZE, BLOCK_DIM_SIZE);

    //if(useGPU){
        const dim3 gridDims(data.width()/blockDims.x+1, data.height()/blockDims.y+1);

        Array2D<Point3<float>>* pointsGPU = points.toGPU();
        BVH* bvhGPU = bvh.toGPU();
        Array2D<float>* dataGPU = data.toGPU();
        const uint sharedMem = (raysPerPoint+1)*sizeof(float);//+64*sizeof(BVHNode); 
        renderGPU<<<gridDims, blockDims, sharedMem>>>(*dataGPU, *pointsGPU, *bvhGPU, raysPerPoint, randomState);
        syncGPU();
        data.fromGPU(dataGPU);
        bvh.fromGPU(bvhGPU);
        points.fromGPU(pointsGPU);
    /*}else{
        int* traceBuffer = (int*) allocMemory(width*height*traceBufferSizePerThread, sizeof(int), useGPU);
        float progress = 0;
        float nextProgress = 0.1;
        #pragma omp parallel for
        for(int y=0; y<height; y++){
            for(int x=0; x<width; x++){
                //render(data, y*width+x, raysPerPoint, points, *bvh, traceBuffer, traceBufferSizePerThread);
                const Point3<float> origin  = points[index];
                Vec3<float> direction = Vec3<float>(0,0,0);
                float result = 0;
                for(uint i=0; i<raysPerPoint; i++){
                    const uint segmentNumber = i%NB_STRATIFIED_DIRS;
                    const float rnd1 = uniform0_1(genEngine);
                    const float rnd2 = uniform0_1(genEngine);
                    const float cosThetaOverPdf = direction.setRandomInHemisphereCosineHost( NB_STRATIFIED_DIRS, segmentNumber, rnd1, rnd2);
                    result += cosThetaOverPdf*bvh.getLighting(origin, direction, &traceBuffer[index*traceBufferSize]);
                }
                data[index] = result/(PI*(float)raysPerPoint); // Diffuse BSDF : f = 1/PI
            }

            #pragma omp atomic
            progress++;
            if(progress >= nextProgress*height){
                nextProgress += 0.1;
            }
        }
        freeMemory(traceBuffer, useGPU);
    }*/
}
