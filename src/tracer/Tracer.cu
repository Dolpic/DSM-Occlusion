#include "hip/hip_runtime.h"
#include "Tracer.cuh"
#include "../utils/utils.cuh"

#include <random>

std::default_random_engine genEngine;
std::uniform_real_distribution<> uniform0_1 = std::uniform_real_distribution<>(0.001, 1);

constexpr unsigned int NB_STRATIFIED_DIRS = 32;
constexpr unsigned int SEED = 1423;
constexpr dim3 threads(8,8);


__host__ __device__ 
void initRender(float* data, Point3<float>* points, Point3<float>** pointsPointers, BVH<float>& bvh, float pixelSize, const unsigned int width, const unsigned int height){
    for(unsigned int y=0; y<height; y++){
        for(unsigned int x=0; x<width; x++){
            const int index = y*width+x;
            points[index] = Point3<float>((float)x*pixelSize,(float)y*pixelSize, data[index]);
            pointsPointers[index] = &(points[index]);
        }
    }
    Array<Point3<float>*> pointsPointersArray = Array<Point3<float>*>(pointsPointers, width*height);
    bvh.build(pointsPointersArray);
}

__global__
void initRenderGPU(float* data, Point3<float>* points, Point3<float>** pointsPointers, BVH<float> bvh, float pixelSize, const unsigned int width, const unsigned int height, hiprandState* const randomState) {
    const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x>=width || y>=height) return;
    const unsigned int index = y*width + x;
    hiprand_init(SEED, index, 0, &randomState[index]);
    initRender(data, points, pointsPointers, bvh, pixelSize, width, height);
}


__host__ 
void render(
    float* const data, const unsigned int index, const unsigned int raysPerPoint, 
    Point3<float>* points, BVH<float>* bvh, BVHNode<float>** traceBuffer, 
    const unsigned int traceBufferSizePerThread){
    
    Point3<float> origin  = points[index];
    Vec3<float> direction = Vec3<float>(0,0,0);
    Ray<float> ray        = Ray<float>(origin, direction);

    float result = 0;
    for(unsigned int i=0; i<raysPerPoint; i++){
        const unsigned int segmentNumber = i%NB_STRATIFIED_DIRS;
        const float rnd1 = uniform0_1(genEngine);
        const float rnd2 = uniform0_1(genEngine);
        const float cosThetaOverPdf = ray.getDirection().setRandomInHemisphereCosine( NB_STRATIFIED_DIRS , segmentNumber, rnd1, rnd2);
        result += cosThetaOverPdf*bvh->getLighting(ray, &traceBuffer[index*traceBufferSizePerThread]);
    }
    data[index] = (result/raysPerPoint)*(1/PI); // Diffuse BSDF
}

__global__
void renderGPU(float* data, Point3<float>* points, int width, int height, BVH<float> bvh, const int raysPerPoint, hiprandState* randomState, BVHNode<float>** traceBuffer, int traceBufferSize){
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x>=width || y>=height) return;

    const unsigned int index = y*width + x;
    Point3<float> origin  = points[index];
    Vec3<float> direction = Vec3<float>(0,0,0);
    Ray<float> ray        = Ray<float>(origin, direction);

    float result = 0;
    for(unsigned int i=0; i<raysPerPoint; i++){
        const float rnd1 = hiprand_uniform(randomState);
        const float rnd2 = hiprand_uniform(randomState);
        const unsigned int segmentNumber = i%NB_STRATIFIED_DIRS;
        const float cosThetaOverPdf = ray.getDirection().setRandomInHemisphereCosine( NB_STRATIFIED_DIRS , segmentNumber, rnd1, rnd2);
        result += cosThetaOverPdf*bvh.getLighting(ray, &traceBuffer[index*traceBufferSize]);
    }
    data[index] = (result/raysPerPoint)*(1/PI); // Diffuse BSDF
}


Tracer::Tracer(float* const data, const bool useGPU, const unsigned int width, const unsigned int height, const float pixelSize) : 
    data(data), useGPU(useGPU), width(width), height(height), pixelSize(pixelSize), bvh(BVH<float>(useGPU, width*height)){
    if(useGPU) randomState = (hiprandState*) allocGPU(width*height, sizeof(hiprandState));
    points = (Point3<float>*) allocMemory(width*height, sizeof(Point3<float>), useGPU);
}

Tracer::~Tracer(){
    if(useGPU) hipFree(randomState);
    freeMemory(points, useGPU);
}

void Tracer::init(const bool prinInfos){
    Point3<float>** pointsArray = (Point3<float>**) allocMemory(width*height, sizeof(Point3<float>*), useGPU); //TODO free after use

    if(useGPU){
        initRenderGPU<<<1,1>>>(data, points, pointsArray, bvh, pixelSize, width, height, randomState);
        checkError(hipGetLastError());
        checkError(hipDeviceSynchronize());
    }else{
        initRender(data, points, pointsArray, bvh, pixelSize, width, height);
        if(prinInfos) bvh.printInfos();
    }
    bvh.freeMemoryAfterBuild();
}


void Tracer::trace(const unsigned int raysPerPoint){
    const int traceBufferSizePerThread = std::log2(bvh.size())+1;
    BVHNode<float>** traceBuffer = (BVHNode<float>**) allocMemory(width*height*traceBufferSizePerThread, sizeof(BVHNode<float>*), useGPU);

    if(useGPU){
        const dim3 blocks(width/threads.x+1, height/threads.y+1);
        renderGPU<<<blocks, threads>>>(data, points, width, height, bvh, raysPerPoint, randomState, traceBuffer, traceBufferSizePerThread);
        checkError(hipGetLastError());
        checkError(hipDeviceSynchronize());
    }else{
        float progress = 0;
        float nextProgress = 0.1;

        #pragma omp parallel for
        for(int y=0; y<height; y++){
            for(int x=0; x<width; x++){
                render(data, y*width+x, raysPerPoint, points, &bvh, traceBuffer, traceBufferSizePerThread);
            }

            #pragma omp atomic
            progress++;
            if(progress >= nextProgress*height){
                std::cout << "Progress " << 100*nextProgress << "%\n";
                std::flush(std::cout);
                nextProgress += 0.1;
            }
        }
    }
    freeMemory(traceBuffer, useGPU);
}
